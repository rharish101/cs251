
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))
#define USAGE_EXIT(s) do\
{\
    printf("Usage: %s <# of elements> <# of threads> \n%s\n", argv[0], s);\
    exit(-1);\
}while(0);

struct num_array
{
    double num1;
    double num2;
    double result;
};

__device__ void function(struct num_array *a)
{
    double square = a ->num1 * a->num1 +  a->num2 * a->num2  + 2 * a->num1 * a->num2;
    a->result = log(square)/sin(square);
    return;
}
__global__ void calculate(char *mem, int num)
{
    int i = blockDim.x * blockDim.y * blockIdx.x + threadIdx.x * blockDim.y + threadIdx.y;
    if(i >= num)
        return;
    struct num_array *a = (struct num_array *)(mem + (i * 3 * sizeof(double)));
    function(a);
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i;
    struct num_array *pa;
    char *ptr;
    char *sptr;
    char *gpu_mem;   
    unsigned long num;   /*Default value of num from MACRO*/
    int blocks, rows, cols;

    if(argc != 4)
        USAGE_EXIT("Not enough parameters");

    num = atoi(argv[1]);   /*Update after checking*/
    if(num <= 0)
        USAGE_EXIT("Invalid number of elements");

    rows = atoi(argv[2]);
    if(rows <= 0)
        USAGE_EXIT("Invalid number of rows");

    cols = atoi(argv[3]);
    if(cols <= 0)
        USAGE_EXIT("Invalid number of columns");
    if (rows * cols > 1024)
        USAGE_EXIT("rows * cols > 1024")

    /* Allocate host (CPU) memory and initialize*/

    ptr = (char *)malloc(num * 3 * sizeof(double));
    sptr = ptr; 
    for(i=0; i<num; ++i)
    {
        pa = (struct num_array *) sptr;
        pa->num1 = (double) i + (double) i * 0.1;
        pa->num2 = pa->num1 + 1.0;
        sptr += 3 * sizeof(double);
    }
    
    
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/
    hipMalloc(&gpu_mem, num * 3 * sizeof(double));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num * 3 * sizeof(double) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
    
    blocks = num / (rows * cols);
    if(num % (rows * cols))
           ++blocks;
    dim3 threads (rows, cols);

    calculate<<<blocks, threads>>>(gpu_mem, num);
    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);
    
    /* Copy back result*/

    hipMemcpy(ptr, gpu_mem, num * 3 * sizeof(double) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
    sptr = ptr;
   
    /*Print the last element for sanity check*/ 
    pa = (struct num_array *) (sptr + (num -1)*3*sizeof(double));
    printf("num1=%f num2=%f result=%f\n", pa->num1, pa->num2, pa->result);

    
    free(ptr);
}
