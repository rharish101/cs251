
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))
#define USAGE_EXIT(s) do\
{\
    printf("Usage: %s <# of elements> <random seed> \n%s\n", argv[0], s);\
    exit(-1);\
}while(0);

__global__ void xor_piece(int *arr, int *step, int num)
{
    int i = (blockDim.x * blockIdx.x + threadIdx.x);
    if (((float) num) / i < *step)
        return;
    i *= *step;
    if ((i >= num) || ((i + (*step) / 2) >= num))
        return;
    arr[i] ^= arr[i + (*step) / 2];
}

__global__ void double_step(int* step)
{
    *step *= 2;
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i;
    int *host_mem;
    int *gpu_mem;
    int *host_step;
    int *gpu_step;
    int *answer;
    unsigned long num;   /*Default value of num from MACRO*/
    int blocks, seed;

    if(argc != 3)
        USAGE_EXIT("Not enough parameters");

    num = atoi(argv[1]);   /*Update after checking*/
    if(num <= 0)
        USAGE_EXIT("Invalid number of elements");

    seed = atoi(argv[2]);   /*Update after checking*/
    if(seed <= 0)
        USAGE_EXIT("Invalid number of elements");

    /* Allocate host (CPU) memory and initialize*/
    host_mem = (int*)malloc(num * sizeof(int));
    srand(seed);
    for(i=0; i<num; ++i){
       host_mem[i] = random();
    }
    answer = (int*)malloc(sizeof(int));
    host_step = (int*)malloc(sizeof(int));
    *host_step = 2;
    
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/
    hipMalloc(&gpu_mem, num * sizeof(int));
    CUDA_ERROR_EXIT("hipMalloc");
    hipMalloc(&gpu_step, sizeof(int));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, host_mem, num * sizeof(int) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    hipMemcpy(gpu_step, host_step, sizeof(int) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
    
    blocks = num / 2048;
    if(num % 2048)
        ++blocks;

    while((*host_step / 2) <= num)
    {
        xor_piece<<<blocks, 1024>>>(gpu_mem, gpu_step, num);
        CUDA_ERROR_EXIT("kernel invocation");

        double_step<<<1, 1>>>(gpu_step);
        CUDA_ERROR_EXIT("kernel invocation");
        *host_step *= 2;
    }

    gettimeofday(&end, NULL);
    
    /* Copy back result*/
    hipMemcpy(answer, gpu_mem, sizeof(int) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");

    gettimeofday(&t_end, NULL);
    
    printf("Total time = %ld microsecs. Processsing = %ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
   
    /*Print the answer*/ 
    printf("Result = %d\n", *answer);

    /**answer = 0;
    for (i = 0; i < num; i++)
        *answer ^= host_mem[i];
    printf("Actual answer = %d\n", *answer);*/
    
    free(host_mem);
    free(host_step);
    free(answer);
}
